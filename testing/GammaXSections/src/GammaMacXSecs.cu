#include "hip/hip_runtime.h"

#include "Declaration.hh"

#include "G4HepEmData.hh"
#include "G4HepEmGammaData.hh"

#include "G4HepEmGammaManager.hh"

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"

// Pull in implementation
#include "G4HepEmGammaManager.icc"
#include "G4HepEmRunUtils.icc"

 __global__
 void TestMacXSecDataKernel ( const struct G4HepEmGammaData* theGammaData_d,
                              int* tsInImat_d, double* tsInEkin_d, double* tsInLogEkin_d,
                              double* tsOutRes_d, int iprocess, int numTestCases) {
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numTestCases; i += blockDim.x * gridDim.x) {
     tsOutRes_d[i] = G4HepEmGammaManager::GetMacXSec (theGammaData_d, tsInImat_d[i], tsInEkin_d[i], tsInLogEkin_d[i], iprocess);
   }
 }

void TestMacXSecDataOnDevice ( const struct G4HepEmData* hepEmData, int* tsInImat_h,
     double* tsInEkinConv_h, double* tsInLogEkinConv_h, double* tsInEkinComp_h, double* tsInLogEkinComp_h,
     double* tsInEkinGNuc_h, double* tsInLogEkinGNuc_h, double* tsOutMXConv_h, double* tsOutMXComp_h,
     double* tsOutMXGNuc_h, int numTestCases ) {
  //
  // --- Allocate device side memory for the input/output data and copy all input
  //     data from host to device
  int*            tsInImat_d = nullptr;
  double*     tsInEkinConv_d = nullptr;
  double*  tsInLogEkinConv_d = nullptr;
  double*     tsInEkinComp_d = nullptr;
  double*  tsInLogEkinComp_d = nullptr;
  double*     tsInEkinGNuc_d = nullptr;
  double*  tsInLogEkinGNuc_d = nullptr;
  double*      tsOutMXConv_d = nullptr;
  double*      tsOutMXComp_d = nullptr;
  double*      tsOutMXGNuc_d = nullptr;

  //
  gpuErrchk ( hipMalloc ( &tsInImat_d,        sizeof( int )    * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkinConv_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkinConv_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkinComp_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkinComp_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkinGNuc_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkinGNuc_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutMXConv_d,     sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutMXComp_d,     sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutMXGNuc_d,     sizeof( double ) * numTestCases ) );
  //
  // --- Copy the input data from host to device (test material index, ekin and log-ekin arrays)
  gpuErrchk ( hipMemcpy ( tsInImat_d,        tsInImat_h,        sizeof( int )    * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkinConv_d,    tsInEkinConv_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkinConv_d, tsInLogEkinConv_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkinComp_d,    tsInEkinComp_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkinComp_d, tsInLogEkinComp_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkinGNuc_d,    tsInEkinGNuc_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkinGNuc_d, tsInLogEkinGNuc_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );

  //
  // --- Launch the kernels
  int numThreads = 512;
  int numBlocks  = std::ceil( float(numTestCases)/numThreads );
  //  std::cout << " N = " << numTestCases << " numBlocks = " << numBlocks << " numThreads = " << numThreads << " x = " << numBlocks*numThreads << std::endl;
  const G4HepEmGammaData* theGammaData_d = hepEmData->fTheGammaData_gpu;
  // conversion
  TestMacXSecDataKernel <<< numBlocks, numThreads >>> (theGammaData_d, tsInImat_d, tsInEkinConv_d, tsInLogEkinConv_d, tsOutMXConv_d, 0,  numTestCases );
  // Compton scatteirng
  TestMacXSecDataKernel <<< numBlocks, numThreads >>> (theGammaData_d, tsInImat_d, tsInEkinComp_d, tsInLogEkinComp_d, tsOutMXComp_d, 1, numTestCases );
  // Gamma-nuclear
  TestMacXSecDataKernel <<< numBlocks, numThreads >>> (theGammaData_d, tsInImat_d, tsInEkinGNuc_d, tsInLogEkinGNuc_d, tsOutMXGNuc_d, 2, numTestCases );
  //
  // --- Synchronize to make sure that completed on the device
  hipDeviceSynchronize();
  //
  // --- Copy the results from the device to the host
  gpuErrchk ( hipMemcpy ( tsOutMXConv_h,     tsOutMXConv_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  gpuErrchk ( hipMemcpy ( tsOutMXComp_h,     tsOutMXComp_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  gpuErrchk ( hipMemcpy ( tsOutMXGNuc_h,     tsOutMXGNuc_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );

  //
  // --- Free all dynamically allocated (device side) memory
  hipFree ( tsInImat_d        );
  hipFree ( tsInEkinConv_d    );
  hipFree ( tsInLogEkinConv_d );
  hipFree ( tsInEkinComp_d    );
  hipFree ( tsInLogEkinComp_d );
  hipFree ( tsInEkinGNuc_d    );
  hipFree ( tsInLogEkinGNuc_d );
  hipFree ( tsOutMXConv_d    );
  hipFree ( tsOutMXComp_d    );
  hipFree ( tsOutMXGNuc_d    );
}
