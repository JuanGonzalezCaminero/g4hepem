#include "G4HepEmElementData.hh"
//#include <iostream>

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"


void CopyElementDataToGPU(struct G4HepEmElementData* onCPU, struct G4HepEmElementData** onGPU) {
  // clean away previous (if any)
  if ( *onGPU ) {
    FreeElementDataOnGPU ( onGPU );
  }
  // allocate array of G4HepEmElemData structures on _d (its pointer adress will on _h)  
  struct G4HepEmElemData* arrayHto_d;
  gpuErrchk ( hipMalloc ( &arrayHto_d, sizeof( struct G4HepEmElemData )*onCPU->fMaxZet ) );
  // - copy the array of G4HepEmElemData structures from the _h to _d
  gpuErrchk ( hipMemcpy ( arrayHto_d, onCPU->fElementData, onCPU->fMaxZet*sizeof( struct G4HepEmElemData ), hipMemcpyHostToDevice ) );
  // now create a helper G4HepEmElementData and set its fMaxZet and 
  // `struct G4HepEmElemData* fElementData` array member, then copy to the 
  // corresponding structure 
  struct G4HepEmElementData* elData_h = new G4HepEmElementData;
  elData_h->fMaxZet      = onCPU->fMaxZet;
  elData_h->fElementData = arrayHto_d;
  gpuErrchk ( hipMalloc ( onGPU, sizeof( struct G4HepEmElementData ) ) );
  gpuErrchk ( hipMemcpy ( *onGPU, elData_h, sizeof( struct G4HepEmElementData ), hipMemcpyHostToDevice ) );
  // Free the auxilary G4HepEmElementData object
  delete elData_h;
}

void FreeElementDataOnGPU ( struct G4HepEmElementData** onGPU ) {  
  if ( *onGPU ) {
    // copy the struct G4HepEmElementData` struct, including its `struct G4HepEmElemData* fElementData` 
    // pointer member, from _d to _h in order to be able to free the _d sice memory 
    // pointed by `fElementData` by calling to hipFree from the host.
    struct G4HepEmElementData* elData_h = new G4HepEmElementData;
    gpuErrchk ( hipMemcpy ( elData_h, *onGPU, sizeof( struct G4HepEmElementData ), hipMemcpyDeviceToHost ) );
    hipFree( elData_h->fElementData );
    // free the whole remaining device side memory (after cleaning all dynamically 
    // allocated members)
    hipFree( *onGPU );
    *onGPU = nullptr;
  }
}
