#include "G4HepEmSBTableData.hh"
//#include <iostream>

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"


void CopySBTableDataToDevice(struct G4HepEmSBTableData* onHOST, struct G4HepEmSBTableData** onDEVICE) {
  if ( !onHOST ) return;
  // clean away previous (if any)
  if ( *onDEVICE ) {
    FreeSBTableDataOnDevice ( onDEVICE );
  }
  // Create a G4HepEmSBTableData structure on the host to store pointers to _d
  // side arrays on the _h side.
  struct G4HepEmSBTableData* sbTablesHTo_d = new G4HepEmSBTableData;
  //
  // set member values
  sbTablesHTo_d->fLogMinElEnergy    = onHOST->fLogMinElEnergy;
  sbTablesHTo_d->fILDeltaElEnergy   = onHOST->fILDeltaElEnergy;
  const int numHepEmMatCuts         = onHOST->fNumHepEmMatCuts;
  const int numElemsInMatCuts       = onHOST->fNumElemsInMatCuts;
  const int numSBTableData          = onHOST->fNumSBTableData;
  sbTablesHTo_d->fNumHepEmMatCuts   = numHepEmMatCuts;
  sbTablesHTo_d->fNumElemsInMatCuts = numElemsInMatCuts;
  sbTablesHTo_d->fNumSBTableData    = numSBTableData;
  //
  // allocate device side memory for the dynamic arrys
  gpuErrchk ( hipMalloc ( &(sbTablesHTo_d->fGammaCutIndxStartIndexPerMC), sizeof( int )    * numHepEmMatCuts   ) );
  gpuErrchk ( hipMalloc ( &(sbTablesHTo_d->fGammaCutIndices),             sizeof( int )    * numElemsInMatCuts ) );
  gpuErrchk ( hipMalloc ( &(sbTablesHTo_d->fSBTableData),                 sizeof( double ) * numSBTableData    ) );
  //
  gpuErrchk ( hipMemcpy (   sbTablesHTo_d->fGammaCutIndxStartIndexPerMC,  onHOST->fGammaCutIndxStartIndexPerMC, sizeof( int )    * numHepEmMatCuts,   hipMemcpyHostToDevice ) );
  gpuErrchk ( hipMemcpy (   sbTablesHTo_d->fGammaCutIndices,              onHOST->fGammaCutIndices,             sizeof( int )    * numElemsInMatCuts, hipMemcpyHostToDevice ) );
  gpuErrchk ( hipMemcpy (   sbTablesHTo_d->fSBTableData,                  onHOST->fSBTableData,                 sizeof( double ) * numSBTableData ,   hipMemcpyHostToDevice ) );
  //
  // Finaly copy the top level, i.e. the main struct with the already
  // appropriate pointers to device side memory locations but stored on the host
  gpuErrchk ( hipMalloc (  onDEVICE,                sizeof(  struct G4HepEmSBTableData ) ) );
  gpuErrchk ( hipMemcpy ( *onDEVICE, sbTablesHTo_d, sizeof(  struct G4HepEmSBTableData ), hipMemcpyHostToDevice ) );
  // and clean
  delete sbTablesHTo_d;
}


void FreeSBTableDataOnDevice(struct G4HepEmSBTableData** onDEVICE) {
  if (*onDEVICE) {
    // copy the on-device data back to host in order to be able to free the device
    // side dynamically allocated memories
    struct G4HepEmSBTableData* onHostTo_d = new G4HepEmSBTableData;
    gpuErrchk ( hipMemcpy( onHostTo_d, *onDEVICE, sizeof( struct G4HepEmSBTableData ), hipMemcpyDeviceToHost ) );
    // ELoss data
    hipFree( onHostTo_d->fGammaCutIndxStartIndexPerMC );
    hipFree( onHostTo_d->fGammaCutIndices             );
    hipFree( onHostTo_d->fSBTableData                 );
    //
    // free the remaining device side electron data and set the host side ptr to null
    hipFree( *onDEVICE );
    *onDEVICE = nullptr;

    delete onHostTo_d;
  }
}
